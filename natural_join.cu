//
// Created by arsho
//

#include <hip/hip_runtime.h>
#include <cstdio>
#include <string>
#include <iostream>
#include <ctime>
#include <ratio>
#include <chrono>
#include <math.h>

using namespace std;

void show_relation(int *data, int total_records,
                   int total_columns, const char *relation_name,
                   int visible_records, int skip_zero) {
    int count = 0;
    cout << "Relation name: " << relation_name << endl;
    cout << "===================================" << endl;
    for (int i = 0; i < total_records; i++) {
        int skip = 0;
        for (int j = 0; j < total_columns; j++) {
            if ((skip_zero == 1) && (data[(i * total_columns) + j] == 0)) {
                skip = 1;
                continue;
            }
            cout << data[(i * total_columns) + j] << " ";
        }
        if (skip == 1)
            continue;
        cout << endl;
        count++;
        if (count == visible_records) {
            cout << "Result cropped at record " << count << endl;
            return;
        }

    }
    cout << "" << endl;
}

void write_relation_to_file(int *data, int total_records, int total_columns, const char *file_name, char separator) {
    FILE *data_file = fopen(file_name, "w");
    for (int i = 0; i < total_records; i++) {
        int skip = 0;
        for (int j = 0; j < total_columns; j++) {
            if (data[(i * total_columns) + j] == 0) {
                skip = 1;
                continue;
            }
            if (j != (total_columns - 1)) {
                fprintf(data_file, "%d%c", data[(i * total_columns) + j], separator);
            } else {
                fprintf(data_file, "%d", data[(i * total_columns) + j]);
            }
        }
        if (skip == 1)
            continue;
        fprintf(data_file, "\n");
    }
    cout << "\nWrote join result to file " << file_name << "\n" << endl;
}


int *get_relation_from_file(const char *file_path, int total_records, int total_columns, char separator) {
    int *data = (int *) malloc(total_records * total_columns * sizeof(int));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_records; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
    return data;
}


int *get_reverse_relation(int *data, int total_records, int total_columns) {
    int *reverse_data = (int *) malloc(total_records * total_columns * sizeof(int));
    for (int i = 0; i < total_records; i++) {
        int pos = total_columns - 1;
        for (int j = 0; j < total_columns; j++) {
            reverse_data[(i * total_columns) + j] = data[(i * total_columns) + pos];
            pos--;
        }
    }
    return reverse_data;
}


__global__
void gpu_get_join_data(int *data, int per_thread_allocation,
                       int *relation_1, int relation_1_records, int relation_1_columns, int relation_1_index,
                       int *relation_2, int relation_2_records, int relation_2_columns, int relation_2_index) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int relation_1_index_value, relation_2_index_value;
    relation_1_index_value = relation_1[(i * relation_1_columns) + relation_1_index];
    int offset = i * per_thread_allocation;
    for (int j = 0; j < relation_2_records; j++) {
        relation_2_index_value = relation_2[(j * relation_2_columns) + relation_2_index];
        if (relation_1_index_value == relation_2_index_value) {
            for (int k = 0; k < relation_1_columns; k++) {
                data[offset++] = relation_1[(i * relation_1_columns) + k];
            }
            for (int k = 0; k < relation_2_columns; k++) {
                if (k != relation_2_index) {
                    data[offset++] = relation_2[(j * relation_2_columns) + k];
                }
            }
        }
    }
}

__global__
void gpu_get_join_size_per_thread(int *join_size,
                                  int *relation_1, int relation_1_records, int relation_1_columns,
                                  int relation_1_index,
                                  int *relation_2, int relation_2_records, int relation_2_columns,
                                  int relation_2_index) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int total_columns = relation_1_columns + relation_2_columns - 1;
    int count = 0;
    int relation_1_index_value, relation_2_index_value;
    relation_1_index_value = relation_1[(i * relation_1_columns) + relation_1_index];
    for (int j = 0; j < relation_2_records; j++) {
        relation_2_index_value = relation_2[(j * relation_2_columns) + relation_2_index];
        if (relation_1_index_value == relation_2_index_value) {
            count += total_columns;
        }
    }
    join_size[i] = count;
}

__global__
void gpu_get_join_data_dynamic(int *data, int *offsets,
                               int *relation_1, int relation_1_records, int relation_1_columns, int relation_1_index,
                               int *relation_2, int relation_2_records, int relation_2_columns, int relation_2_index) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int relation_1_index_value, relation_2_index_value;
    relation_1_index_value = relation_1[(i * relation_1_columns) + relation_1_index];
    int offset = offsets[i];
    for (int j = 0; j < relation_2_records; j++) {
        relation_2_index_value = relation_2[(j * relation_2_columns) + relation_2_index];
        if (relation_1_index_value == relation_2_index_value) {
            for (int k = 0; k < relation_1_columns; k++) {
                data[offset++] = relation_1[(i * relation_1_columns) + k];
            }
            for (int k = 0; k < relation_2_columns; k++) {
                if (k != relation_2_index) {
                    data[offset++] = relation_2[(j * relation_2_columns) + k];
                }
            }
        }
    }
}

void cpu_get_join_data(int *data, long long data_max_length,
                       int *relation_1, int relation_1_records, int relation_1_columns, int relation_1_index,
                       int *relation_2, int relation_2_records, int relation_2_columns, int relation_2_index) {
    long long row_count = 0, column_count = 0;
    int total_columns = relation_1_columns + relation_2_columns - 1;
    int relation_1_index_value, relation_2_index_value;
    for (int i = 0; i < relation_1_records; i++) {
        relation_1_index_value = relation_1[(i * relation_1_columns) + relation_1_index];
        for (int j = 0; j < relation_2_records; j++) {
            relation_2_index_value = relation_2[(j * relation_2_columns) + relation_2_index];
            if (relation_1_index_value == relation_2_index_value) {
                column_count = 0;
                for (int k = 0; k < relation_1_columns; k++) {
                    data[(row_count * total_columns) + column_count] = relation_1[(i * relation_1_columns) + k];
                    column_count++;
                }
                for (int k = 0; k < relation_2_columns; k++) {
                    if (k != relation_2_index) {
                        data[(row_count * total_columns) + column_count] = relation_2[(j * relation_2_columns) + k];
                        column_count++;
                    }
                }
                row_count++;
                if (row_count == data_max_length - 1) {
                    break;
                }
            }
        }
        if (row_count == data_max_length - 1) {
            break;
        }
    }
}

void gpu_join_relations(char *data_path, char separator, char *output_path,
                        int relation_columns, int relation_1_records,
                        int relation_2_records, int total_records, int visible_records) {

    int total_columns = relation_columns + relation_columns - 1;
    int *relation_1_data = get_relation_from_file(data_path,
                                                  relation_1_records, relation_columns,
                                                  separator);
    int *relation_2_data = get_reverse_relation(relation_1_data,
                                                relation_1_records,
                                                relation_columns);
    int *join_result = (int *) malloc(total_records * total_columns * sizeof(int));

    int *gpu_relation_1_data, *gpu_relation_2_data, *gpu_join_result;
    hipMalloc((void **) &gpu_relation_1_data, relation_1_records * relation_columns * sizeof(int));
    hipMalloc((void **) &gpu_relation_2_data, relation_2_records * relation_columns * sizeof(int));
    hipMalloc((void **) &gpu_join_result, total_records * total_columns * sizeof(int));

    hipMemcpy(gpu_relation_1_data, relation_1_data, relation_1_records * relation_columns * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(gpu_relation_2_data, relation_2_data, relation_2_records * relation_columns * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(gpu_join_result, join_result, total_records * total_columns * sizeof(int), hipMemcpyHostToDevice);

//    dim3 grid_size = (1, 1);
//    dim3 block_size = 10;

    int block_size = sqrt(relation_1_records);
    int grid_size = sqrt(relation_1_records);
    int per_thread_allocation = (total_records * total_columns) / (block_size * grid_size);


    gpu_get_join_data<<<grid_size, block_size>>>(gpu_join_result, per_thread_allocation,
                                                 gpu_relation_1_data, relation_1_records,
                                                 relation_columns, 0,
                                                 gpu_relation_2_data, relation_2_records,
                                                 relation_columns, 0);

    hipDeviceSynchronize();
    hipMemcpy(join_result, gpu_join_result, total_records * total_columns * sizeof(int), hipMemcpyDeviceToHost);

    show_relation(relation_1_data, relation_1_records, relation_columns,
                  "Relation 1", visible_records, 1);
    show_relation(relation_2_data, relation_2_records, relation_columns,
                  "Relation 2", visible_records, 1);
    show_relation(join_result, total_records,
                  total_columns, "GPU Join Result", visible_records, 1);
    write_relation_to_file(join_result, total_records, total_columns,
                           output_path, separator);
    hipFree(gpu_relation_1_data);
    hipFree(gpu_relation_2_data);
    hipFree(gpu_join_result);
    free(join_result);
    free(relation_1_data);
    free(relation_2_data);

}

void gpu_join_relations_2_pass(char *data_path, char separator, char *output_path,
                               int relation_1_records, int relation_1_columns,
                               int relation_2_records, int relation_2_columns, int visible_records) {
    int total_columns = relation_1_columns + relation_2_columns - 1;
    int block_size = sqrt(relation_1_records);
    int grid_size = sqrt(relation_1_records);
    int *relation_1 = get_relation_from_file(data_path,
                                             relation_1_records, relation_1_columns,
                                             separator);
    int *relation_2 = get_reverse_relation(relation_1,
                                           relation_1_records,
                                           relation_2_columns);
    int *join_size_per_thread = (int *) malloc(relation_1_records * sizeof(int));
    int *offset = (int *) malloc(relation_1_records * sizeof(int));
    int *gpu_relation_1, *gpu_relation_2, *gpu_join_size_per_thread, *gpu_offset, *gpu_join_result;
    hipMalloc((void **) &gpu_relation_1, relation_1_records * relation_1_columns * sizeof(int));
    hipMalloc((void **) &gpu_relation_2, relation_2_records * relation_2_columns * sizeof(int));
    hipMalloc((void **) &gpu_join_size_per_thread, relation_1_records * sizeof(int));
    hipMalloc((void **) &gpu_offset, relation_1_records * sizeof(int));
    hipMemcpy(gpu_relation_1, relation_1, relation_1_records * relation_1_columns * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(gpu_relation_2, relation_2, relation_2_records * relation_2_columns * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(gpu_join_size_per_thread, join_size_per_thread, relation_1_records * sizeof(int),
               hipMemcpyHostToDevice);
    gpu_get_join_size_per_thread<<<grid_size, block_size>>>(gpu_join_size_per_thread,
                                                            gpu_relation_1, relation_1_records,
                                                            relation_1_columns, 0,
                                                            gpu_relation_2, relation_2_records,
                                                            relation_2_columns, 0);
    hipDeviceSynchronize();
    hipMemcpy(join_size_per_thread, gpu_join_size_per_thread, relation_1_records * sizeof(int),
               hipMemcpyDeviceToHost);
    int total_size = join_size_per_thread[0];
    for (int i = 1; i < relation_1_records; i++) {
        offset[i] = offset[i - 1] + join_size_per_thread[i - 1];
        total_size += join_size_per_thread[i];
    }
    hipMemcpy(gpu_offset, offset, relation_1_records * sizeof(int), hipMemcpyHostToDevice);
    int *join_result = (int *) malloc(total_size * sizeof(int));
    hipMalloc((void **) &gpu_join_result, total_size * sizeof(int));
    hipMemcpy(gpu_join_result, join_result, total_size * sizeof(int), hipMemcpyHostToDevice);

    gpu_get_join_data_dynamic<<<grid_size, block_size>>>(gpu_join_result, gpu_offset,
                                                         gpu_relation_1, relation_1_records,
                                                         relation_1_columns, 0,
                                                         gpu_relation_2, relation_2_records,
                                                         relation_2_columns, 0);
    hipDeviceSynchronize();
    hipMemcpy(join_result, gpu_join_result, total_size * sizeof(int), hipMemcpyDeviceToHost);

    show_relation(relation_1, relation_1_records, relation_1_columns,
                  "Relation 1", visible_records, 1);
    show_relation(relation_2, relation_2_records, relation_2_columns,
                  "Relation 2", visible_records, 1);
    show_relation(join_result, total_size / 3,
                  total_columns, "GPU Join Result", visible_records, 0);
    write_relation_to_file(join_result, total_size / 3, total_columns,
                           output_path, separator);
    hipFree(gpu_relation_1);
    hipFree(gpu_relation_2);
    hipFree(gpu_join_size_per_thread);
    free(join_size_per_thread);
    free(relation_1);
    free(relation_2);
}


void cpu_join_relations(char *data_path, char separator, char *output_path,
                        int relation_1_records, int relation_1_columns,
                        int relation_2_records, int relation_2_columns,
                        int total_records, int visible_records) {
    int total_columns = relation_1_columns + relation_2_columns - 1;
    int *relation_1_data = get_relation_from_file(data_path,
                                                  relation_1_records, relation_1_columns,
                                                  separator);
    int *relation_2_data = get_reverse_relation(relation_1_data,
                                                relation_1_records,
                                                relation_2_columns);
    int *join_result = (int *) malloc(total_records * total_columns * sizeof(int));
    show_relation(relation_1_data, relation_1_records, relation_1_columns,
                  "Relation 1", visible_records, 1);
    show_relation(relation_2_data, relation_2_records, relation_2_columns,
                  "Relation 2", visible_records, 1);
    cpu_get_join_data(join_result, total_records, relation_1_data, relation_1_records,
                      relation_1_columns, 0,
                      relation_2_data, relation_2_records,
                      relation_2_columns, 0);
    show_relation(join_result, total_records,
                  total_columns, "CPU Join Result", visible_records, 1);
    write_relation_to_file(join_result, total_records, total_columns,
                           output_path, separator);
    free(relation_1_data);
    free(relation_2_data);
    free(join_result);
}

int main() {
    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
    char *data_path, *output_path;
    char separator = '\t';
    int relation_1_records, relation_1_columns, relation_2_records, relation_2_columns,
            total_records, visible_records;

//    data_path = "data/link.facts_412148.txt";
//    output_path = "output/join_medium_cpu_262144.txt";
//    relation_1_records = 262144;
//    relation_2_records = 262144;
//    total_records = relation_1_records * 5;
//    relation_1_columns = 2;
//    relation_2_columns = 2;
//    visible_records = 10;
//    cpu_join_relations(data_path, separator, output_path, relation_1_records, relation_1_columns,
//                       relation_2_records, relation_2_columns, total_records, visible_records);


    data_path = "data/link.facts_412148.txt";
    output_path = "output/join_medium_gpu_262144_dynamic.txt";
    relation_1_records = 262144;
    relation_2_records = 262144;
    relation_1_columns = 2;
    relation_2_columns = 2;
    visible_records = 10;
    gpu_join_relations_2_pass(data_path, separator, output_path,
                              relation_1_records, relation_1_columns,
                              relation_2_records, relation_2_columns, visible_records);
    // 262144

    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> time_span = t2 - t1;

    cout << "\nTotal time: " << time_span.count() << " seconds\n" << endl;
    return 0;
}