#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"


using namespace std;

void gpu_tc(const char *data_path, char separator,
            long int relation_rows, double load_factor,
            int preferred_grid_size, int preferred_block_size, const char *dataset_name, int number_of_sm) {
    int relation_columns = 2;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    std::chrono::high_resolution_clock::time_point temp_time_begin;
    std::chrono::high_resolution_clock::time_point temp_time_end;
    KernelTimer timer;
    time_point_begin = chrono::high_resolution_clock::now();
    double spent_time;
    output.initialization_time = 0;
    output.join_time = 0;
    output.projection_time = 0;
    output.deduplication_time = 0;
    output.memory_clear_time = 0;
    output.union_time = 0;
    output.total_time = 0;
    double sort_time = 0.0;
    double unique_time = 0.0;
    double merge_time = 0.0;
    double temp_spent_time = 0.0;

    int block_size, grid_size;
    int *relation;
    int *relation_host;
    Entity *hash_table, *result, *t_delta;
    Entity *result_host;
    long int join_result_rows;
    long int t_delta_rows = relation_rows;
    long int result_rows = relation_rows;
    long int iterations = 0;
    long int hash_table_rows = (long int) relation_rows / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));

    checkCuda(hipHostMalloc((void **) &relation_host, relation_rows * relation_columns * sizeof(int)));
    checkCuda(hipMalloc((void **) &relation, relation_rows * relation_columns * sizeof(int)));
    checkCuda(hipMalloc((void **) &result, result_rows * sizeof(Entity)));
    checkCuda(hipMalloc((void **) &t_delta, relation_rows * sizeof(Entity)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));
//    checkCuda(hipMemPrefetchAsync(relation, relation_rows * relation_columns * sizeof(int), device_id));
    block_size = 512;
    grid_size = 32 * number_of_sm;
    if (preferred_grid_size != 0) {
        grid_size = preferred_grid_size;
    }
    if (preferred_block_size != 0) {
        block_size = preferred_block_size;
    }
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.initialization_time += spent_time;
    time_point_begin = chrono::high_resolution_clock::now();
    get_relation_from_file_gpu(relation_host, data_path,
                               relation_rows, relation_columns, separator);
    hipMemcpy(relation, relation_host, relation_rows * relation_columns * sizeof(int),
               hipMemcpyHostToDevice);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.read_time = spent_time;

    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    time_point_begin = chrono::high_resolution_clock::now();
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.initialization_time += spent_time;
    timer.start_timer();
    build_hash_table<<<grid_size, block_size>>>
            (hash_table, hash_table_rows,
             relation, relation_rows,
             relation_columns);
    checkCuda(hipDeviceSynchronize());
    timer.stop_timer();
    spent_time = timer.get_spent_time();
//    cout << "Hash table build time: " << spent_time << endl;
    output.hashtable_build_time = spent_time;
    output.hashtable_build_rate = (double) relation_rows / spent_time;
    output.join_time += spent_time;

    timer.start_timer();
    // initial result and t delta both are same as the input relation
    initialize_result_t_delta<<<grid_size, block_size>>>(result, t_delta, relation, relation_rows, relation_columns);
    checkCuda(hipDeviceSynchronize());
    timer.stop_timer();
    spent_time = timer.get_spent_time();
    output.union_time += spent_time;
    temp_time_begin = chrono::high_resolution_clock::now();
    thrust::stable_sort(thrust::device, result, result + relation_rows, cmp());
    temp_time_end = chrono::high_resolution_clock::now();
    temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
    sort_time += temp_spent_time;
    output.deduplication_time += temp_spent_time;

    time_point_begin = chrono::high_resolution_clock::now();
    hipFree(relation);
    hipHostFree(relation_host);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.memory_clear_time += spent_time;

#ifdef DEBUG
    cout << "| Iteration | #T Delta | #Join | #Deduplicated join | #Union | #Deduplicated Union | ";
    cout << "Join(s) | Union(s) | Deduplication(s) | Memory clear(s)|"<< endl;
    cout << "| --- | --- | --- | --- | --- | --- | --- | --- | --- | --- |" << endl;
#endif
    while (true) {
        double temp_join = 0.0, temp_union = 0.0, temp_deduplication = 0.0, temp_memory_clear = 0.0;
        double temp_merge = 0.0, temp_sort = 0.0, temp_unique = 0.0;
#ifdef DEBUG
        cout << "| " << iterations+1 << " | "<< t_delta_rows << " | ";
#endif
        time_point_begin = chrono::high_resolution_clock::now();
        int *offset;
        Entity *join_result;
        checkCuda(hipMalloc((void **) &offset, t_delta_rows * sizeof(int)));
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join += spent_time;
        output.join_time += spent_time;
        timer.start_timer();
        get_join_result_size<<<grid_size, block_size>>>(hash_table, hash_table_rows, t_delta, t_delta_rows,
                                                        offset);
        checkCuda(hipDeviceSynchronize());
        timer.stop_timer();
        spent_time = timer.get_spent_time();
        temp_join += spent_time;
        output.join_time += spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        join_result_rows = thrust::reduce(thrust::device, offset, offset + t_delta_rows, 0);
        thrust::exclusive_scan(thrust::device, offset, offset + t_delta_rows, offset);
        checkCuda(hipMalloc((void **) &join_result, join_result_rows * sizeof(Entity)));
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join += spent_time;
        output.join_time += spent_time;
        timer.start_timer();
        get_join_result<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                   t_delta, t_delta_rows, offset, join_result);
        checkCuda(hipDeviceSynchronize());
        timer.stop_timer();
        spent_time = timer.get_spent_time();
        temp_join += spent_time;
        output.join_time += spent_time;
#ifdef DEBUG
        cout << join_result_rows << " | ";
#endif
        // deduplication of projection
        // first sort the array and then remove consecutive duplicated elements
        temp_time_begin = chrono::high_resolution_clock::now();
        thrust::stable_sort(thrust::device, join_result, join_result + join_result_rows, cmp());
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_sort += temp_spent_time;
        temp_deduplication += temp_spent_time;
        sort_time += temp_spent_time;
        output.deduplication_time += temp_spent_time;
        temp_time_begin = chrono::high_resolution_clock::now();
        long int projection_rows = (thrust::unique(thrust::device,
                                                   join_result, join_result + join_result_rows,
                                                   is_equal())) - join_result;
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_unique += temp_spent_time;
        temp_deduplication += temp_spent_time;
        unique_time += temp_spent_time;
        output.deduplication_time += temp_spent_time;
#ifdef DEBUG
        cout << projection_rows << " | ";
#endif
        // show_entity_array(join_result, projection_rows, "join_result");
        time_point_begin = chrono::high_resolution_clock::now();
        hipFree(t_delta);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_memory_clear += spent_time;
        output.memory_clear_time += spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        checkCuda(hipMalloc((void **) &t_delta, projection_rows * sizeof(Entity)));
        thrust::copy(thrust::device, join_result, join_result + projection_rows, t_delta);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join += spent_time;
        output.join_time += spent_time;

        time_point_begin = chrono::high_resolution_clock::now();
        Entity *concatenated_result;
        long int concatenated_rows = projection_rows + result_rows;
        checkCuda(hipMalloc((void **) &concatenated_result, concatenated_rows * sizeof(Entity)));
        temp_time_begin = chrono::high_resolution_clock::now();
        // merge two sorted array: previous result and join result
        thrust::merge(thrust::device,
                      result, result + result_rows,
                      join_result, join_result + projection_rows,
                      concatenated_result, cmp());
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_merge += temp_spent_time;
        merge_time += temp_spent_time;
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_union += spent_time;
        output.union_time += spent_time;
#ifdef DEBUG
        cout << concatenated_rows << " | ";
#endif
        long int deduplicated_result_rows;
        temp_time_begin = chrono::high_resolution_clock::now();
        deduplicated_result_rows = (thrust::unique(thrust::device,
                                                   concatenated_result,
                                                   concatenated_result + concatenated_rows,
                                                   is_equal())) - concatenated_result;
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_unique += temp_spent_time;
        unique_time += temp_spent_time;
        temp_deduplication += temp_spent_time;
        output.deduplication_time += temp_spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        hipFree(result);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_memory_clear += spent_time;
        output.memory_clear_time += spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        checkCuda(hipMalloc((void **) &result, deduplicated_result_rows * sizeof(Entity)));
        // Copy the deduplicated concatenated result to result
        thrust::copy(thrust::device, concatenated_result,
                     concatenated_result + deduplicated_result_rows, result);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_union += spent_time;
        output.union_time += spent_time; // changed this time from deduplication to union
#ifdef DEBUG
        cout << deduplicated_result_rows << " | ";
#endif
        t_delta_rows = projection_rows;
        time_point_begin = chrono::high_resolution_clock::now();
        hipFree(join_result);
        hipFree(offset);
        hipFree(concatenated_result);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_memory_clear += spent_time;
        output.memory_clear_time += spent_time;
#ifdef DEBUG
        cout << temp_join << " | ";
        cout << temp_union << " | ";
        cout << temp_deduplication << " | ";
        cout << temp_memory_clear << " | " << endl;
#endif

        if (result_rows == deduplicated_result_rows) {
            iterations++;
            break;
        }
        result_rows = deduplicated_result_rows;
        iterations++;
    }
//    show_entity_array(result, result_rows, "Result");
    time_point_begin = chrono::high_resolution_clock::now();
    checkCuda(hipHostMalloc((void **) &result_host, result_rows * sizeof(Entity)));
    hipMemcpy(result_host, result, result_rows * sizeof(Entity),
               hipMemcpyDeviceToHost);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.union_time += spent_time;
    time_point_begin = chrono::high_resolution_clock::now();
    hipFree(t_delta);
    hipFree(result);
    hipFree(hash_table);
    hipHostFree(result_host);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.memory_clear_time += spent_time;
    double calculated_time = output.initialization_time +
                             output.read_time + output.reverse_time + output.hashtable_build_time + output.join_time +
                             output.projection_time +
                             output.union_time + output.deduplication_time + output.memory_clear_time;
    cout << endl;
    cout << "| Dataset | Number of rows | TC size | Iterations | Blocks x Threads | Time (s) |" << endl;
    cout << "| --- | --- | --- | --- | --- | --- |" << endl;
    cout << "| " << dataset_name << " | " << relation_rows << " | " << result_rows;
    cout << fixed << " | " << iterations << " | ";
    cout << fixed << grid_size << " x " << block_size << " | " << calculated_time << " |\n" << endl;
    output.block_size = block_size;
    output.grid_size = grid_size;
    output.input_rows = relation_rows;
    output.load_factor = load_factor;
    output.hashtable_rows = hash_table_rows;
    output.dataset_name = dataset_name;
    output.total_time = calculated_time;

    cout << endl;
    cout << "Initialization: " << output.initialization_time;
    cout << ", Read: " << output.read_time << endl;
    cout << "Hashtable rate: " << output.hashtable_build_rate << " keys/s, time: ";
    cout << output.hashtable_build_time << endl;
    cout << "Join: " << output.join_time << endl;
    cout << "Deduplication: " << output.deduplication_time;
    cout << " (sort: " << sort_time << ", unique: " << unique_time << ")" << endl;
    cout << "Memory clear: " << output.memory_clear_time << endl;
    cout << "Union: " << output.union_time << " (merge: " << merge_time << ")" << endl;
    cout << "Total: " << output.total_time << endl;
}


void run_benchmark(int grid_size, int block_size, double load_factor) {
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    std::locale loc("");
    std::cout.imbue(loc);
    std::cout << std::fixed;
    std::cout << std::setprecision(4);
    char separator = '\t';
    string datasets[] = {
            "OL.cedge_initial", "../data/data_7035.txt",
            "CA-HepTh", "../data/data_51971.txt",
            "SF.cedge", "../data/data_223001.txt",
            "ego-Facebook", "../data/data_88234.txt",
            "wiki-Vote", "../data/data_103689.txt",
            "p2p-Gnutella09", "../data/data_26013.txt",
            "p2p-Gnutella04", "../data/data_39994.txt",
            "cal.cedge", "../data/data_21693.txt",
            "TG.cedge", "../data/data_23874.txt",
            "OL.cedge", "../data/data_7035.txt",
            "luxembourg_osm", "../data/data_119666.txt",
            "fe_sphere", "../data/data_49152.txt",
            "fe_body", "../data/data_163734.txt",
            "cti", "../data/data_48232.txt",
            "fe_ocean", "../data/data_409593.txt",
            "wing", "../data/data_121544.txt",
            "loc-Brightkite", "../data/data_214078.txt",
            "delaunay_n16", "../data/data_196575.txt",
            "usroads", "../data/data_165435.txt",
            "usroads-48", "../data/data_161950.txt",
            "String 9990", "../data/data_9990.txt",
            "String 2990", "../data/data_2990.txt",
            "string 4", "../data/data_4.txt",
            "talk 5", "../data/data_5.txt",
            "cyclic 3", "../data/data_3.txt",
    };
    for (int i = 0; i < sizeof(datasets) / sizeof(datasets[0]); i += 2) {
        const char *data_path, *dataset_name;
        dataset_name = datasets[i].c_str();
        data_path = datasets[i + 1].c_str();
        long int row_size = get_row_size(data_path);
        cout << "Benchmark for " << dataset_name << endl;
        cout << "----------------------------------------------------------" << endl;
        gpu_tc(data_path, separator,
               row_size, load_factor,
               grid_size, block_size, dataset_name, number_of_sm);
        cout << endl;

    }
}

int main() {
    run_benchmark(0, 0, 0.1);
    return 0;
}

// Benchmark
// nvcc tc_cuda.cu -run -o tc_cuda.out
// or
// make run
