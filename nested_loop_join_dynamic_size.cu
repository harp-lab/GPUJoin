#include "hip/hip_runtime.h"
//
// Created by arsho
//
#include <cstdio>
#include <string>
#include <iostream>
#include <chrono>
#include <math.h>
#include <assert.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include "utils.h"

using namespace std;

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}


__global__
void gpu_get_join_size_per_thread(int *join_size_per_thread,
                                  int *relation_1, int relation_1_rows, int relation_1_columns,
                                  int relation_1_index,
                                  int *relation_2, int relation_2_rows, int relation_2_columns,
                                  int relation_2_index) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= relation_1_rows) return;
    int total_columns = relation_1_columns + relation_2_columns - 1;
    int count = 0;
    int relation_1_index_value, relation_2_index_value;
    relation_1_index_value = relation_1[(i * relation_1_columns) + relation_1_index];
    for (int j = 0; j < relation_2_rows; j++) {
        relation_2_index_value = relation_2[(j * relation_2_columns) + relation_2_index];
        if (relation_1_index_value == relation_2_index_value) {
            count += total_columns;
        }
    }
    join_size_per_thread[i] = count;
}

__global__
void gpu_get_join_data_dynamic(int *result, int *offsets,
                               int *relation_1, int relation_1_rows, int relation_1_columns, int relation_1_index,
                               int *relation_2, int relation_2_rows, int relation_2_columns, int relation_2_index) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= relation_1_rows) return;
    int relation_1_index_value, relation_2_index_value;
    relation_1_index_value = relation_1[(i * relation_1_columns) + relation_1_index];
    int offset = offsets[i];
    for (int j = 0; j < relation_2_rows; j++) {
        relation_2_index_value = relation_2[(j * relation_2_columns) + relation_2_index];
        if (relation_1_index_value == relation_2_index_value) {
            for (int k = 0; k < relation_1_columns; k++) {
                result[offset++] = relation_1[(i * relation_1_columns) + k];
            }
            for (int k = 0; k < relation_2_columns; k++) {
                if (k != relation_2_index) {
                    result[offset++] = relation_2[(j * relation_2_columns) + k];
                }
            }
        }
    }
}


void gpu_join_relations_2_pass(const char *data_path, char separator, const char *output_path,
                               int relation_1_rows, int relation_1_columns,
                               int relation_2_rows, int relation_2_columns) {
    double total_time, pass_1_time, pass_2_time, offset_time;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    std::chrono::duration<double> time_span;
    int total_columns = relation_1_columns + relation_2_columns - 1;
    int threads_per_block, blocks_per_grid;
    threads_per_block = 1024;
    blocks_per_grid = ceil((double) relation_1_rows / threads_per_block);
    cout << "GPU join operation (non-atomic): ";
    cout << "(" << relation_1_rows << ", " << relation_1_columns << ")";
    cout << " x (" << relation_2_rows << ", " << relation_2_columns << ")" << endl;
    cout << "Blocks per grid: " << blocks_per_grid;
    cout << ", Threads per block: " << threads_per_block << endl;
    time_point_begin = chrono::high_resolution_clock::now();
    int *gpu_relation_1, *gpu_relation_2, *gpu_offset, *gpu_join_result;
    checkCuda(hipMallocManaged(&gpu_relation_1, relation_1_rows * relation_1_columns * sizeof(int)));
    checkCuda(hipMallocManaged(&gpu_relation_2, relation_2_rows * relation_2_columns * sizeof(int)));
    checkCuda(hipMallocManaged(&gpu_offset, relation_1_rows * sizeof(int)));

    get_relation_from_file_gpu(gpu_relation_1, data_path,
                               relation_1_rows, relation_1_columns,
                               separator);
    get_reverse_relation_gpu(gpu_relation_2, gpu_relation_1,
                             relation_1_rows,
                             relation_2_columns);

    time_point_begin = chrono::high_resolution_clock::now();
    gpu_get_join_size_per_thread<<<blocks_per_grid, threads_per_block>>>(gpu_offset,
                                                                         gpu_relation_1, relation_1_rows,
                                                                         relation_1_columns, 0,
                                                                         gpu_relation_2, relation_2_rows,
                                                                         relation_2_columns, 0);
    checkCuda(hipDeviceSynchronize());
    time_point_end = chrono::high_resolution_clock::now();
    time_span = time_point_end - time_point_begin;
    pass_1_time = time_span.count();
    show_time_spent("GPU Pass 1 get join size per row in relation 1", time_point_begin, time_point_end);
    time_point_begin = chrono::high_resolution_clock::now();
    int total_size = thrust::reduce(thrust::device, gpu_offset, gpu_offset + relation_1_rows, 0);
    thrust::exclusive_scan(thrust::device, gpu_offset, gpu_offset + relation_1_rows, gpu_offset);

    cout << "Total size of the join result: " << total_size << endl;
    time_point_end = chrono::high_resolution_clock::now();
    time_span = time_point_end - time_point_begin;
    offset_time = time_span.count();
    show_time_spent("Thrust calculate offset", time_point_begin, time_point_end);
    checkCuda(hipMallocManaged(&gpu_join_result, total_size * sizeof(int)));
    time_point_begin = chrono::high_resolution_clock::now();
    gpu_get_join_data_dynamic<<<blocks_per_grid, threads_per_block>>>(gpu_join_result, gpu_offset,
                                                                      gpu_relation_1, relation_1_rows,
                                                                      relation_1_columns, 0,
                                                                      gpu_relation_2, relation_2_rows,
                                                                      relation_2_columns, 0);
    checkCuda(hipDeviceSynchronize());
    time_point_end = chrono::high_resolution_clock::now();
    time_span = time_point_end - time_point_begin;
    pass_2_time = time_span.count();
    show_time_spent("GPU Pass 2 join operation", time_point_begin, time_point_end);
//    time_point_begin = chrono::high_resolution_clock::now();
//    write_relation_to_file(gpu_join_result, total_size / 3, total_columns,
//                           output_path, separator);
//    time_point_end = chrono::high_resolution_clock::now();
//    show_time_spent("Write result", time_point_begin, time_point_end);
    total_time = pass_1_time + offset_time + pass_2_time;
    cout << "Total time (pass 1 + offset + pass 2): " << total_time << endl;
    cout << "| Number of rows | #Blocks | #Threads | #Result rows | Pass 1 | Offset calculation | Pass 2 | Total time |"
         << endl;
    cout << "| " << relation_1_rows << " | " << blocks_per_grid << " | " << threads_per_block << " | ";
    cout << total_size / 3 << " | " << pass_1_time << " | " << offset_time << " | " << pass_2_time << " | ";
    cout << total_time << " |" << endl;
    hipFree(gpu_relation_1);
    hipFree(gpu_relation_2);
    hipFree(gpu_offset);
}

int main() {

    char separator = '\t';
    int relation_1_rows, relation_1_columns, relation_2_rows, relation_2_columns;
    relation_1_columns = 2;
    relation_2_columns = 2;

//    relation_1_rows = 412148;
//    relation_2_rows = 412148;
//    data_path = "data/link.facts_412148.txt";
//    output_path = "output/join_gpu_412148_atomic.txt";

    relation_1_rows = 500000;
    relation_2_rows = 500000;
    const char *data_path = "data/data_500000.txt";
    const char *output_path = "output/join_gpu_500000.txt";
    gpu_join_relations_2_pass(data_path, separator, output_path,
                              relation_1_rows, relation_1_columns,
                              relation_2_rows, relation_2_columns);
    cout << endl;

//    int n = 100000;
//    int increment = 50000;
//    int count = 0;
//
//    while (count < 19) {
//        relation_1_rows = n;
//        relation_2_rows = n;
//        string a = "data/data_" + std::to_string(n) + ".txt";
//        string b = "output/join_gpu_" + std::to_string(n) + ".txt";
//        const char *data_path = a.c_str();
//        const char *output_path = b.c_str();
//
//        gpu_join_relations_2_pass(data_path, separator, output_path,
//                                  relation_1_rows, relation_1_columns,
//                                  relation_2_rows, relation_2_columns);
//
//        cout << endl;
//        n += increment;
//        count++;
//    }

    return 0;
}