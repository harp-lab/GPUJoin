#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include "utils.h"


using namespace std;

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

struct Entity {
    int key;
    int value;
};

struct Output {
    int block_size;
    int grid_size;
    long int key_size;
    long int hashtable_rows;
    double load_factor;
    int duplicate_percentage;
    double build_time;
    long int build_rate;
    double join_pass_1;
    double join_offset;
    double join_pass_2;
    long int join_rows;
    long int join_columns;
    double total_time;
} output;


struct is_match_gpu {
    int key;

    __host__ __device__ is_match_gpu(int searched_key) : key(searched_key) {};

    __device__
    bool operator()(Entity &x) {
        return x.key == key;
    }
};

/*
 * Method that returns position in the hashtable for a key using Murmur3 hash
 * */
__device__ int get_position(int key, int hash_table_row_size) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
    return key & (hash_table_row_size - 1);
}

__global__
void build_hash_table(Entity *hash_table, long int hash_table_row_size,
                      int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = relation[(i * relation_columns) + 0];
        int value = relation[(i * relation_columns) + 1];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            int existing_key = atomicCAS(&hash_table[position].key, 0, key);
            if (existing_key == 0) {
                hash_table[position].value = value;
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}

__global__
void get_join_result_size(Entity *hash_table, long int hash_table_row_size,
                          int *reverse_relation, long int relation_rows, int relation_columns,
                          int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = reverse_relation[(i * relation_columns) + 0];
//        int current_size = thrust::count_if(thrust::device, hash_table,
//                                            hash_table + hash_table_row_size, is_match_gpu(key));
        int current_size = 0;
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == 0) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }

        current_size = current_size * 3;
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result(Entity *hash_table, int hash_table_row_size,
                     int *reverse_relation, int relation_rows, int relation_columns, int *offset, int *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < relation_rows; i += stride) {
        int key = reverse_relation[(i * relation_columns) + 0];
        int value = reverse_relation[(i * relation_columns) + 1];
        int start_index = offset[i];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index++] = key;
                join_result[start_index++] = hash_table[position].value;
                join_result[start_index++] = value;
            } else if (hash_table[position].key == 0) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}


void gpu_hashjoin(const char *data_path, char separator,
                  long int relation_rows, int relation_columns, double load_factor, int max_duplicate_percentage,
                  int preferred_grid_size, int preferred_block_size, const char *output_path) {
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Added to display comma separated integer values
    std::locale loc("");
    std::cout.imbue(loc);

    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, min_grid_size, grid_size;
    double spent_time;
    int *relation, *reverse_relation, *join_result, *offset;
    Entity *hash_table;
    const char *random_datapath = "random";
    long int hash_table_size, join_result_size, join_result_rows, relation_size;
    int join_result_columns = (relation_columns * 2) - 1;


    int hash_table_row_size = (int) relation_rows / load_factor;
    hash_table_row_size = pow(2, ceil(log(hash_table_row_size) / log(2)));

    relation_size = relation_rows * relation_columns * sizeof(int);
    hash_table_size = hash_table_row_size * sizeof(Entity);

    checkCuda(hipMallocManaged(&relation, relation_size));
    checkCuda(hipMallocManaged(&reverse_relation, relation_size));
    checkCuda(hipMallocManaged(&offset, relation_rows * sizeof(int)));
    checkCuda(hipMallocManaged(&hash_table, hash_table_size));
    checkCuda(hipMemPrefetchAsync(relation, relation_size, device_id));

    if (strcmp(data_path, random_datapath) == 0) {
        generate_random_relation(relation, relation_rows, relation_columns, max_duplicate_percentage);
    } else {
        get_relation_from_file_gpu(relation, data_path,
                                   relation_rows, relation_columns, separator);
    }
    get_reverse_relation_gpu(reverse_relation, relation,
                             relation_rows,
                             relation_columns);
    checkCuda(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                                                 build_hash_table, 0, 0));
    grid_size = 32 * number_of_sm;
    if ((preferred_grid_size != 0) && (preferred_block_size != 0)) {
        grid_size = preferred_grid_size;
        block_size = preferred_block_size;
    }
    output.block_size = block_size;
    output.grid_size = grid_size;
    output.key_size = relation_rows;
    output.load_factor = load_factor;
    output.hashtable_rows = hash_table_row_size;
    output.load_factor = load_factor;
    output.duplicate_percentage = max_duplicate_percentage;
    checkCuda(hipEventRecord(start));
    build_hash_table<<<grid_size, block_size>>>
            (hash_table, hash_table_row_size,
             relation, relation_rows,
             relation_columns);
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);
    double gpu_time_s = gpu_time / 1000.0f;
    long int rate = relation_rows / gpu_time_s;
    output.build_time = gpu_time_s;
    output.build_rate = rate;

    checkCuda(hipEventRecord(start));
    get_join_result_size<<<grid_size, block_size>>>
            (hash_table, hash_table_row_size,
             reverse_relation, relation_rows,
             relation_columns, offset);
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);
    gpu_time_s = gpu_time / 1000.0f;
    time_point_begin = chrono::high_resolution_clock::now();
    join_result_size = thrust::reduce(thrust::device, offset, offset + relation_rows, 0);
    thrust::exclusive_scan(thrust::device, offset, offset + relation_rows, offset);
    join_result_rows = join_result_size / join_result_columns;
    output.join_rows = join_result_rows;
    output.join_columns = join_result_columns;
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.join_pass_1 = gpu_time_s;
    output.join_offset = spent_time;
    cout << "Join result: " << join_result_rows << " x " << join_result_columns << endl;
    checkCuda(hipMallocManaged(&join_result, join_result_size * sizeof(int)));
    checkCuda(hipEventRecord(start));
    get_join_result<<<grid_size, block_size>>>
            (hash_table, hash_table_row_size,
             reverse_relation, relation_rows,
             relation_columns, offset, join_result);
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);
    gpu_time_s = gpu_time / 1000.0f;
    output.join_pass_2 = gpu_time_s;
    write_relation_to_file(join_result, join_result_rows, join_result_columns,
                           output_path, separator);
    output.total_time = output.build_time + output.join_pass_1 + output.join_offset + output.join_pass_2;
    hipFree(relation);
    hipFree(reverse_relation);
    hipFree(hash_table);
    hipFree(join_result);
    hipFree(offset);
    std::cout << std::fixed;
    std::cout << std::setprecision(4);
    cout << "| #Input | #Join | #BlocksXThreads | #Hashtable | Load factor | Duplicate ";
    cout << "| Build rate | Total(Build+Pass 1+Offset+Pass 2) |" << endl;
    cout << "| " << output.key_size << " | " << output.join_rows;
    cout << " | " << output.grid_size << "X" << output.block_size << " | ";
    cout << output.hashtable_rows << " | " << output.load_factor << " | ";
    if (strcmp(data_path, random_datapath) == 0) {
        cout << output.duplicate_percentage << " | ";
    } else {
        cout << "N/A | ";
    }
    cout << fixed << output.build_rate << " | ";
    cout << fixed << output.total_time;
    cout << fixed << " (" << output.build_time << "+" << output.join_pass_1 << "+";
    cout << fixed << output.join_offset << "+" << output.join_pass_2 << ") |\n" << endl;
}

/**
 * Main function to create a hashtable on an input relation, reverse it, and join the original relation with reverse one
 * The parameters are given as sequential command line arguments.
 *
 * @args: Data path, Relation rows, Relation columns, Load factor, Max duplicate percentage, Grid size, Block size
 * Data path: (filepath or random) (string)
 * Load factor: 0 - 1 (double)
 * Max duplicate percentage: 0-99 (int), will not be used if data path is not random
 * Grid size: 0 for predefined value based on number of SMs of the GPU
 * Block size: 0 for predefined value based on occupancy API
 * @return 0
 */
int main(int argc, char **argv) {
    const char *data_path, *output_path;
    char separator = '\t';
    int relation_rows, relation_columns, max_duplicate_percentage, grid_size, block_size;
    double load_factor;
    data_path = argv[1];
    if (sscanf(argv[2], "%i", &relation_rows) != 1) {
        fprintf(stderr, "error - not an integer");
    }
    if (sscanf(argv[3], "%i", &relation_columns) != 1) {
        fprintf(stderr, "error - not an integer");
    }
    if (sscanf(argv[4], "%lf", &load_factor) != 1) {
        fprintf(stderr, "error - not a double");
    }
    if (sscanf(argv[5], "%i", &max_duplicate_percentage) != 1) {
        fprintf(stderr, "error - not an integer");
    }
    if (sscanf(argv[6], "%i", &grid_size) != 1) {
        fprintf(stderr, "error - not an integer");
    }
    if (sscanf(argv[7], "%i", &block_size) != 1) {
        fprintf(stderr, "error - not an integer");
    }
    output_path = "output/gpu_hj.txt";
    gpu_hashjoin(data_path, separator,
                 relation_rows, relation_columns, load_factor, max_duplicate_percentage,
                 grid_size, block_size, output_path);
    return 0;
}

// Parameters: Data path, Relation rows, Relation columns, Load factor, Max duplicate percentage, Grid size, Block size

// nvcc hashjoin_gpu.cu -run -o join -run-args data/link.facts_412148.txt -run-args 25000 -run-args 2 -run-args 0.3 -run-args 30 -run-args 0 -run-args 0
// nvcc hashjoin_gpu.cu -run -o join -run-args random -run-args 25000 -run-args 2 -run-args 0.3 -run-args 30 -run-args 0 -run-args 0

// nvcc hashjoin_gpu.cu -run -o join -run-args data/link.facts_412148.txt -run-args 412148 -run-args 2 -run-args 0.3 -run-args 30 -run-args 0 -run-args 0
// nvcc hashjoin_gpu.cu -run -o join -run-args random -run-args 1000000 -run-args 2 -run-args 0.3 -run-args 30 -run-args 0 -run-args 0
