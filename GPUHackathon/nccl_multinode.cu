#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>


#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

// String hashing function
static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}

// Use unistd to access system host name, then change period
// to null to make string
static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}


int main(int argc, char* argv[])
{
  // Test size, could be anything
  int size = 32*1024*1024;

  // CPU rank, total CPU ranks, local rank is for keeping track when 
  // overprescribing threads per cpu
  int myRank, nRanks, localRank = 0;


  // Initialize MPI variables
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));


  // Find hash for each CPU host and use allgather to send array of all hashes to each thread
  // If multiple threads on one CPU device, use localRank to keep track
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  /* int MPI_Allgather(const void *sendbuf, int  sendcount,
        MPI_Datatype sendtype, void *recvbuf, int recvcount,
        MPI_Datatype recvtype, MPI_Comm comm) */
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }


  // Use two GPUs for each MPI CPU thread
  int nDev = 2;

  // Create buffers and CUDA streams for each GPU
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  // Creating and filling buffers and streams on GPUs, have to use localrank
  // if a CPU has multiple MPI threads
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(localRank*nDev + i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }

  // Create NCCL communicator for the GPUs on this CPU
  ncclUniqueId id;
  ncclComm_t comms[nDev];

  // Generate NCCL unique ID at root process and broadcast it to all
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  // Initialize NCCL, group API is required around ncclCommInitRank as it is
  // called across multiple GPUs in each thread/process
  NCCLCHECK(ncclGroupStart());
  for (int i=0; i<nDev; i++) {
    CUDACHECK(hipSetDevice(localRank*nDev + i));
    // ncclResult_t ncclCommInitRank(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank)
    // The rank of each GPU device is MPI rank * GPUs per MPI thread + i
    NCCLCHECK(ncclCommInitRank(comms+i, nRanks*nDev, id, myRank*nDev + i));
  }
  NCCLCHECK(ncclGroupEnd());


  // Use NCCL group API to call allReduce on every GPU
  NCCLCHECK(ncclGroupStart());
  for (int i=0; i<nDev; i++)
    // ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream)
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
           comms[i], s[i]));
   NCCLCHECK(ncclGroupEnd());


  // Synchronize on CUDA stream to complete NCCL communication
  for (int i=0; i<nDev; i++)
      CUDACHECK(hipStreamSynchronize(s[i]));


  // Free device memory
  for (int i=0; i<nDev; i++) {
     CUDACHECK(hipFree(sendbuff[i]));
     CUDACHECK(hipFree(recvbuff[i]));
  }


  // Finalize NCCL
  for (int i=0; i<nDev; i++) {
     ncclCommDestroy(comms[i]);
  }


  // Finalize MPI
  MPICHECK(MPI_Finalize());

  // Print success
  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}