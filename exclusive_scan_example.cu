#include <stdio.h>
#include <iostream>
#include <hipcub/hipcub.hpp>

using namespace std;

int main() {
    int num_items = 7;
    int *d_in;          // e.g., [8, 6, 7, 5, 3, 0, 9]
    int *d_out;         // e.g., [ ,  ,  ,  ,  ,  ,  ]

    size_t size = num_items * sizeof(int);

    hipMallocManaged(&d_in, size);
    hipMallocManaged(&d_out, size);

    for (int i = 0; i < num_items; i++) {
        d_in[i] = i * 2;
    }

// Determine temporary device storage requirements
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
// Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
// Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
    hipDeviceSynchronize();
    // d_out s<-- [0, 8, 14, 21, 26, 29, 29]
    for (int i = 0; i < num_items; i++) {
        cout << i << " " << d_in[i] << " " << d_out[i] << endl;
    }
    cout << endl;

    hipFree(d_in);
    hipFree(d_out);

//// Determine temporary device storage requirements
//    void     *d_temp_storage = NULL;
//    size_t   temp_storage_bytes = 0;
//    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
//// Allocate temporary storage
//    hipMalloc(&d_temp_storage, temp_storage_bytes);
//// Run exclusive prefix sum
//    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
//// d_out s<-- [0, 8, 14, 21, 26, 29, 29]


//    thrust::host_vector<Entity> map_host(1);
//    thrust::device_vector<Entity> map_device = map_host;
//    map_device[0].key = 3;
//    map_device[0].value = 13;
//    cout << map_device[0].key << endl;
//
////    thrust::host_vector<Entity> relation_host(relation_rows);
////    thrust::host_vector<Entity> reverse_relation_host(reverse_relation_rows);
//
//    for (long int i = 0; i < relation_rows; i++) {
//        int key = relation[(i * relation_columns) + 0];
//        int value = relation[(i * relation_columns) + 1];
//        int reverse_key = reverse_relation[(i * relation_columns) + 0];
//        int reverse_value = reverse_relation[(i * relation_columns) + 1];
//        relation_host[i].key = key;
//        relation_host[i].value = value;
//        reverse_relation_host[i].key = reverse_key;
//        reverse_relation_host[i].value = reverse_value;
//    }

//    thrust::device_vector<Entity> relation_device = relation_host;
//    thrust::device_vector<Entity> reverse_relation_device = reverse_relation_host;
//
//
//    cout << "Relation host (Unsorted):" << endl;
//    for (long int i = 0; i < relation_rows; i++) {
//        cout << relation_host[i].key << ", " << relation_host[i].value << endl;
//    }
//
//    thrust::stable_sort(thrust::device, relation_device.begin(), relation_device.end(), cmp());
//
//    relation_host = relation_device;
//
//    cout << "Relation host (Sorted):" << endl;
//    for (long int i = 0; i < relation_rows; i++) {
//        cout << relation_host[i].key << ", " << relation_host[i].value << endl;
//    }

//    Entity *concatenated_result;
//    long int concatenated_rows = 5;
//    checkCuda(hipMallocManaged(&concatenated_result, concatenated_rows * sizeof(Entity)));
//    concatenated_result[0].key = 1;
//    concatenated_result[0].value = 1;
//    concatenated_result[1].key = 1;
//    concatenated_result[1].value = 1;
//    concatenated_result[2].key = 2;
//    concatenated_result[2].value = 2;
//    concatenated_result[3].key = 1;
//    concatenated_result[3].value = 1;
//    concatenated_result[4].key = 2;
//    concatenated_result[4].value = 1;
//    // (1, 1), (1, 1), (2, 2), (1, 1), (2, 1) -> (1, 1), (2, 2), (2, 1)
//
//    thrust::stable_sort(thrust::device, concatenated_result, concatenated_result + concatenated_rows, cmp());
//    cout << "Sorted input: " << endl;
//    for (long int i = 0; i < concatenated_rows; i++) {
//        cout << concatenated_result[i].key << ", " << concatenated_result[i].value << endl;
//    }
//
//    long int deduplicated_result_rows = (thrust::unique(thrust::device,
//                                                        concatenated_result,
//                                                        concatenated_result + concatenated_rows,
//                                                        is_equal())) - concatenated_result;
//    cout << "Deduplicated result: " << endl;
//    for (long int i = 0; i < deduplicated_result_rows; i++) {
//        cout << concatenated_result[i].key << ", " << concatenated_result[i].value << endl;
//    }

    return 0;
}