#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#define N 10000000
#define MAX_ERR 1e-6

__global__ void initBuffer(int *out, int n) {
    for(int i = 0; i < n; i++){
        out[i] = 1;
    }
}

int main(int argc, char* argv[])
{
    //managing 4 devices
    int nDev = 4;
    int size = 32*1024*1024;
    int devs[nDev] = { 0, 1, 2, 3};
    ncclComm_t comms[nDev];
    int check[1];

    for (int i = 0; i < nDev; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    //allocating and initializing device buffers
    int** sendbuff = (int**)malloc(nDev * sizeof(int*));
    int** recvbuff = (int**)malloc(nDev * sizeof(int*));
    hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


    for (int i = 0; i < nDev; ++i) {
        hipSetDevice(i);
        hipMalloc(sendbuff + i, size * sizeof(int));
        hipMalloc(recvbuff + i, size * sizeof(int));
        hipStreamCreate(s+i);
        initBuffer<<<1,1>>>(sendbuff[i], size);
    }


    //initializing NCCL
    ncclCommInitAll(comms, nDev, devs);


    //calling NCCL communication API. Group API is required when using
    //multiple devices per thread
    ncclGroupStart();
    for (int i = 0; i < nDev; ++i)
        ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
                      comms[i], s[i]);
    ncclGroupEnd();


    //synchronizing on CUDA streams to wait for completion of NCCL operation
    for (int i = 0; i < nDev; ++i) {
        hipSetDevice(i);
        hipStreamSynchronize(s[i]);
    }

    hipMemcpy(check, recvbuff[0], sizeof(int), hipMemcpyDeviceToHost);
    printf("Value of check %d\n", check[0]);

    //free device buffers
    for (int i = 0; i < nDev; ++i) {
        hipSetDevice(i);
        hipFree(sendbuff[i]);
        hipFree(recvbuff[i]);
    }


    //finalizing NCCL
    for(int i = 0; i < nDev; ++i)
        ncclCommDestroy(comms[i]);


    printf("Success \n");
    return 0;
}
