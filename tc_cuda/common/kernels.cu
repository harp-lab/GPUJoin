#include "hip/hip_runtime.h"
/*
 * Method that returns position in the hashtable for a key using Murmur3 hash
 * */


__global__
void build_hash_table(Entity *hash_table, long int hash_table_row_size,
                      int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = relation[(i * relation_columns) + 0];
        int value = relation[(i * relation_columns) + 1];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            int existing_key = atomicCAS(&hash_table[position].key, -1, key);
            if (existing_key == -1) {
                hash_table[position].value = value;
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}

__global__
void initialize_t_delta(Entity *t_delta, int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        t_delta[i].key = relation[(i * relation_columns) + 0];
        t_delta[i].value = relation[(i * relation_columns) + 1];
    }
}

__global__
void initialize_result_t_delta(Entity *result, Entity *t_delta,
                       int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        t_delta[i].key = result[i].key = relation[(i * relation_columns) + 0];
        t_delta[i].value = result[i].value = relation[(i * relation_columns) + 1];
    }
}

__global__
void copy_struct(Entity *source, long int source_rows, Entity *destination) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= source_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < source_rows; i += stride) {
        destination[i].key = source[i].key;
        destination[i].value = source[i].value;
    }
}

__global__
void negative_fill_struct(Entity *source, long int source_rows) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= source_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < source_rows; i += stride) {
        source[i].key = -1;
        source[i].value = -1;
    }
}

__global__
void get_reverse_relation(int *relation, long int relation_rows, int relation_columns, Entity *t_delta) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (long int i = index; i < relation_rows; i += stride) {
        t_delta[i].key = relation[(i * relation_columns) + 0];
        t_delta[i].value = relation[(i * relation_columns) + 1];
    }
}


__global__
void get_join_result_size(Entity *hash_table, long int hash_table_row_size,
                          Entity *t_delta, long int relation_rows,
                          int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[i].value;
        int current_size = 0;
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result(Entity *hash_table, int hash_table_row_size,
                     Entity *t_delta, int relation_rows, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[i].value;
        int value = t_delta[i].key;
        int start_index = offset[i];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = value;
                join_result[start_index].value = hash_table[position].value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}
