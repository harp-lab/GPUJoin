#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <iostream>
#include <chrono>
#include <cstdio>
#include <string>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/set_operations.h>
#include "common/error_handler.cu"
#include "common/utils.cu"
#include "common/kernels.cu"


using namespace std;


void gpu_tc(const char *data_path, char separator,
            long int relation_rows, double load_factor,
            int preferred_grid_size, int preferred_block_size, const char *dataset_name, bool benchmark) {
    int lazy_step = 3;
    int relation_columns = 2;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    std::cout << std::fixed;
    std::cout << std::setprecision(4);
    time_point_begin = chrono::high_resolution_clock::now();
    double spent_time;
    output.initialization_time = 0;
    output.join_time = 0;
    output.projection_time = 0;
    output.deduplication_time = 0;
    output.memory_clear_time = 0;
    output.union_time = 0;
    output.total_time = 0;

    // Added to display comma separated integer values
    std::locale loc("");
    std::cout.imbue(loc);
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);
    int block_size, grid_size;
    int *relation, *reverse_relation;
    Entity *hash_table, *result;
    long int join_result_rows;
    long int reverse_relation_rows = relation_rows;
    long int result_rows = relation_rows;
    long int iterations = 0;
    int join_result_columns = relation_columns;
    long int hash_table_rows = (long int) relation_rows / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));
//    cout << "Hash table rows: " << hash_table_rows << endl;

    checkCuda(hipMallocManaged(&relation, relation_rows * relation_columns * sizeof(int)));
    checkCuda(hipMallocManaged(&reverse_relation, reverse_relation_rows * relation_columns * sizeof(int)));
    checkCuda(hipMallocManaged(&result, result_rows * sizeof(Entity)));
    checkCuda(hipMallocManaged(&hash_table, hash_table_rows * sizeof(Entity)));
    checkCuda(hipMemPrefetchAsync(relation, relation_rows * relation_columns * sizeof(int), device_id));
//    checkCuda(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
//                                                 build_hash_table, 0, 0));
    block_size = 512;
    grid_size = 32 * number_of_sm;
    if (preferred_grid_size != 0) {
        grid_size = preferred_grid_size;
    }
    if (preferred_block_size != 0) {
        block_size = preferred_block_size * number_of_sm;
    }
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.initialization_time += spent_time;
    time_point_begin = chrono::high_resolution_clock::now();
    get_relation_from_file_gpu(relation, data_path,
                               relation_rows, relation_columns, separator);

    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.read_time = spent_time;
    time_point_begin = chrono::high_resolution_clock::now();
    get_reverse_relation<<<grid_size, block_size>>>(relation, relation_rows, relation_columns,
                                                    reverse_relation);
    checkCuda(hipDeviceSynchronize());
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.reverse_time = spent_time;

    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    time_point_begin = chrono::high_resolution_clock::now();
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.initialization_time += spent_time;
    time_point_begin = chrono::high_resolution_clock::now();
    build_hash_table<<<grid_size, block_size>>>
            (hash_table, hash_table_rows,
             relation, relation_rows,
             relation_columns);
    checkCuda(hipDeviceSynchronize());
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
//    cout << "Hash table build time: " << spent_time << endl;
    output.hashtable_build_time = spent_time;
    output.hashtable_build_rate = relation_rows / spent_time;
    output.join_time += spent_time;

    time_point_begin = chrono::high_resolution_clock::now();
    // initial result is the input relation
    initialize_result<<<grid_size, block_size>>>
            (result,
             relation, relation_rows,
             relation_columns);
    checkCuda(hipDeviceSynchronize());
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.union_time += spent_time;
    long int previous_unique_result_rows = result_rows;

//    cout
//            << "| Iteration | # Deduplicated join | # Deduplicated union | Join(s) | Deduplication(s) | Projection(s) | Union(s) |"
//            << endl;
//    cout << "| --- | --- | --- | --- | --- | --- | --- |" << endl;
    while (true) {
        double temp_join_time = 0, temp_projection_time = 0, temp_deduplication_time = 0, temp_union_time = 0;
        int *offset;
        Entity *join_result;
        checkCuda(hipMallocManaged(&offset, reverse_relation_rows * sizeof(int)));
        time_point_begin = chrono::high_resolution_clock::now();
        get_join_result_size<<<grid_size, block_size>>>
                (hash_table, hash_table_rows,
                 reverse_relation, reverse_relation_rows,
                 relation_columns, offset);
        checkCuda(hipDeviceSynchronize());
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join_time += spent_time;
        output.join_time += spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        join_result_rows = thrust::reduce(thrust::device, offset, offset + reverse_relation_rows, 0);
        thrust::exclusive_scan(thrust::device, offset, offset + reverse_relation_rows, offset);
        checkCuda(hipMallocManaged(&join_result, join_result_rows * sizeof(Entity)));
        get_join_result<<<grid_size, block_size>>>
                (hash_table, hash_table_rows,
                 reverse_relation, reverse_relation_rows,
                 relation_columns, offset, join_result);
        checkCuda(hipDeviceSynchronize());
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join_time += spent_time;
        output.join_time += spent_time;

        // deduplication of projection
        // first sort the array and then remove consecutive duplicated elements
        time_point_begin = chrono::high_resolution_clock::now();
        thrust::stable_sort(thrust::device, join_result, join_result + join_result_rows,
                            cmp());
        long int projection_rows = (thrust::unique(thrust::device,
                                                   join_result, join_result + join_result_rows,
                                                   is_equal())) - join_result;
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_deduplication_time += spent_time;
        output.deduplication_time += spent_time;

        Entity *projection;
        checkCuda(hipMallocManaged(&projection, projection_rows * sizeof(Entity)));
        checkCuda(hipMallocManaged(&reverse_relation, projection_rows * relation_columns * sizeof(int)));

        time_point_begin = chrono::high_resolution_clock::now();
        get_reverse_projection<<<grid_size, block_size>>>
                (join_result, projection,
                 reverse_relation, projection_rows, join_result_columns);
        checkCuda(hipDeviceSynchronize());
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_projection_time += spent_time;
        output.projection_time += spent_time;
        // concatenated result = result + projection
        time_point_begin = chrono::high_resolution_clock::now();
        Entity *concatenated_result;
        long int concatenated_rows = projection_rows + result_rows;
//        cout << "Concatenated rows: " << concatenated_rows << endl;
        checkCuda(hipMallocManaged(&concatenated_result, concatenated_rows * sizeof(Entity)));
//        concatenated_rows = thrust::set_union(thrust::device, result, result + result_rows, projection, projection+projection_rows, concatenated_result, cmp()) - concatenated_result;
//        cout << "Unique concatenated rows: " << concatenated_rows << endl;
        thrust::copy(thrust::device, result, result + result_rows, concatenated_result);
        thrust::copy(thrust::device, projection, projection + projection_rows,
                     concatenated_result + result_rows);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_union_time += spent_time;
        output.union_time += spent_time;

        // deduplication of projection
        // first sort the array and then remove consecutive duplicated elements
        long int deduplicated_result_rows;
        if (iterations % lazy_step == 0) {
            time_point_begin = chrono::high_resolution_clock::now();
            thrust::stable_sort(thrust::device, concatenated_result, concatenated_result + concatenated_rows,
                                cmp());
            deduplicated_result_rows = (thrust::unique(thrust::device,
                                                       concatenated_result,
                                                       concatenated_result + concatenated_rows,
                                                       is_equal())) - concatenated_result;
            hipFree(result);
            checkCuda(hipMallocManaged(&result, deduplicated_result_rows * sizeof(Entity)));
            // Copy the deduplicated concatenated result to result
            thrust::copy(thrust::device, concatenated_result,
                         concatenated_result + deduplicated_result_rows, result);
            time_point_end = chrono::high_resolution_clock::now();
            spent_time = get_time_spent("", time_point_begin, time_point_end);
            temp_deduplication_time += spent_time;
            output.deduplication_time += spent_time;
        } else {
            time_point_begin = chrono::high_resolution_clock::now();
            hipFree(result);
            checkCuda(hipMallocManaged(&result, concatenated_rows * sizeof(Entity)));
//             Copy the deduplicated concatenated result to result
            thrust::copy(thrust::device, concatenated_result,
                         concatenated_result + concatenated_rows, result);
            time_point_end = chrono::high_resolution_clock::now();
            spent_time = get_time_spent("", time_point_begin, time_point_end);
            output.deduplication_time += spent_time;
        }
        reverse_relation_rows = projection_rows;
//        show_entity_array(concatenated_result, concatenated_rows, "concatenated_result");
        time_point_begin = chrono::high_resolution_clock::now();
        hipFree(join_result);
        hipFree(offset);
        hipFree(projection);
        hipFree(concatenated_result);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        output.memory_clear_time += spent_time;
//        cout << "| " << iterations << " | ";
//        cout << projection_rows << " | " << result_rows << " | ";
//        cout << temp_join_time << " | " << temp_deduplication_time << " | " << temp_projection_time << " | ";
//        cout << temp_union_time << " |" << endl;
        if (iterations % lazy_step == 0) {
            result_rows = deduplicated_result_rows;
            if (previous_unique_result_rows == deduplicated_result_rows) {
                break;
            }
            previous_unique_result_rows = result_rows;
        } else {
            result_rows = concatenated_rows;
        }
//        cout << "Iteration: " << iterations << ", result rows: " << result_rows << endl;
        iterations++;
    }
//    show_entity_array(result, result_rows, "Result");
    time_point_begin = chrono::high_resolution_clock::now();
    hipFree(relation);
    hipFree(reverse_relation);
    hipFree(result);
    hipFree(hash_table);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.memory_clear_time += spent_time;
    double calculated_time = output.initialization_time +
                             output.read_time + output.reverse_time + output.hashtable_build_time + output.join_time +
                             output.projection_time +
                             output.union_time + output.deduplication_time + output.memory_clear_time;
    cout << endl;
    cout << "| Dataset | Number of rows | TC size | Iterations | Blocks x Threads | Time (s) |" << endl;
    cout << "| --- | --- | --- | --- | --- | --- |" << endl;
    cout << "| " << dataset_name << " | " << relation_rows << " | " << result_rows;
    cout << fixed << " | " << iterations << " | ";
    cout << fixed << grid_size << " x " << block_size << " | " << calculated_time << " |\n" << endl;
    output.block_size = block_size;
    output.grid_size = grid_size;
    output.input_rows = relation_rows;
    output.load_factor = load_factor;
    output.hashtable_rows = hash_table_rows;
    output.dataset_name = dataset_name;
    output.total_time = calculated_time;

    cout << endl;
    cout << "Initialization: " << output.initialization_time;
    cout << ", Read: " << output.read_time << ", reverse: " << output.reverse_time << endl;
    cout << "Hashtable rate: " << output.hashtable_build_rate << " keys/s, time: ";
    cout << output.hashtable_build_time << endl;
    cout << "Join: " << output.join_time << endl;
    cout << "Projection: " << output.projection_time << endl;
    cout << "Deduplication: " << output.deduplication_time << endl;
    cout << "Memory clear: " << output.memory_clear_time << endl;
    cout << "Union: " << output.union_time << endl;
    cout << "Total: " << output.total_time << endl;
}


void run_benchmark(int grid_size, int block_size, double load_factor) {
    char separator = '\t';
    string datasets[] = {
//            "SF.cedge", "data/data_223001.txt",
//            "p2p-Gnutella09", "data/data_26013.txt",
//            "p2p-Gnutella04", "data/data_39994.txt",
//            "cal.cedge", "data/data_21693.txt",
//            "TG.cedge", "data/data_23874.txt",
            "OL.cedge", "../data/data_7035.txt",
////            "string 4", "data/data_4.txt",
////            "talk 5", "data/data_5.txt",
////            "cyclic 3", "data/data_3.txt",
//            "string 55555", "data/data_55555.txt",
//            "roadNet-TX", "data/data_3843320.txt"
    };
    for (int i = 0; i < sizeof(datasets) / sizeof(datasets[0]); i += 2) {
        const char *data_path, *dataset_name;
        dataset_name = datasets[i].c_str();
        data_path = datasets[i + 1].c_str();
        long int row_size = get_row_size(data_path);
        cout << "Benchmark for " << dataset_name << endl;
        cout << "----------------------------------------------------------" << endl;
        gpu_tc(data_path, separator,
               row_size, load_factor,
               grid_size, block_size, dataset_name, true);
        cout << endl;

    }
}

int main() {
    run_benchmark(0, 0, 0.1);
    return 0;
}

// Benchmark
// nvcc tc_cuda.cu -run -o tc_cuda.out
// or
// make run
